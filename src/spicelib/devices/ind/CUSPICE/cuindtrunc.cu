#include "hip/hip_runtime.h"
/**********
Copyright 2014 - NGSPICE Software
Author: 2014 Francesco Lannutti
**********/

#include "ngspice/config.h"
#include "CUSPICE/cucktterr.cuh"
#include "inddefs.h"

extern "C"
__global__ void cuINDtrunc_kernel (INDparamGPUstruct, int, double **, double *, double,
                                   int, int, double, double, double, double, double *, int *) ;

extern "C"
int
cuINDtrunc
(
GENmodel *inModel, CKTcircuit *ckt, double *timeStep
)
{
    (void)timeStep ;

    INDmodel *model = (INDmodel *)inModel ;
    int thread_x, thread_y, block_x ;

    hipError_t status ;

    /*  loop through all the inductor models */
    for ( ; model != NULL ; model = model->INDnextModel)
    {
        /* Determining how many blocks should exist in the kernel */
        thread_x = 1 ;
        thread_y = 256 ;
        if (model->n_instances % thread_y != 0)
            block_x = (int)((model->n_instances + thread_y - 1) / thread_y) ;
        else
            block_x = model->n_instances / thread_y ;

        dim3 thread (thread_x, thread_y) ;

        /* Kernel launch */
        status = hipGetLastError () ; // clear error status

        cuINDtrunc_kernel <<< block_x, thread >>> (model->INDparamGPU, model->n_instances,
                                                   ckt->dD_CKTstates, ckt->d_CKTdeltaOld,
                                                   ckt->CKTdelta, ckt->CKTorder, ckt->CKTintegrateMethod,
                                                   ckt->CKTabstol, ckt->CKTreltol, ckt->CKTchgtol, ckt->CKTtrtol,
                                                   ckt->d_CKTtimeSteps, model->d_PositionVector_timeSteps) ;

        hipDeviceSynchronize () ;

        status = hipGetLastError () ; // check for launch error
        if (status != hipSuccess)
        {
            fprintf (stderr, "Kernel launch failure in the Trunc Inductor Model\n\n") ;
            return (E_NOMEM) ;
        }
    }

    return (OK) ;
}

extern "C"
__global__
void
cuINDtrunc_kernel
(
INDparamGPUstruct INDentry, int n_instances, double **CKTstates,
double *CKTdeltaOld, double CKTdelta, int CKTorder, int CKTintegrateMethod,
double CKTabsTol, double CKTrelTol, double CKTchgTol, double CKTtrTol,
double *CKTtimeSteps, int *PositionVector_timeSteps
)
{
    int instance_ID ;

    instance_ID = threadIdx.y + blockDim.y * blockIdx.x ;
    if (instance_ID < n_instances)
    {
        if (threadIdx.x == 0)
        {
            cuCKTterr (INDentry.d_INDstateArray [instance_ID], CKTstates,
                       CKTdeltaOld, CKTdelta, CKTorder, CKTintegrateMethod,
                       CKTabsTol, CKTrelTol, CKTchgTol, CKTtrTol,
                       &(CKTtimeSteps [PositionVector_timeSteps [instance_ID]])) ;
        }
    }

    return ;
}
