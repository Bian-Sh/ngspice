#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014, NVIDIA Corporation. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, 
 * are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, 
 *    this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice, 
 *    this list of conditions and the following disclaimer in the documentation and/or 
 *    other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors may be used to 
 *    endorse or promote products derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, 
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, 
 * OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; 
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, 
 * STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "ngspice/config.h"
#include "ngspice/CUSPICE/cuniinteg.cuh"
#include "inddefs.h"

/* hipMalloc MACRO to check it for errors --> CUDAMALLOCCHECK(name of pointer, dimension, type, status) */
#define CUDAMALLOCCHECK(a, b, c, d) \
    if (d != hipSuccess) \
    { \
        fprintf (stderr, "cuMUTload routine...\n") ; \
        fprintf (stderr, "Error: hipMalloc failed on %s size of %d bytes\n", #a, (int)(b * sizeof(c))) ; \
        fprintf (stderr, "Error: %s = %d, %s\n", #d, d, hipGetErrorString (d)) ; \
        return (E_NOMEM) ; \
    }

/* hipMemcpy MACRO to check it for errors --> CUDAMEMCPYCHECK(name of pointer, dimension, type, status) */
#define CUDAMEMCPYCHECK(a, b, c, d) \
    if (d != hipSuccess) \
    { \
        fprintf (stderr, "cuMUTload routine...\n") ; \
        fprintf (stderr, "Error: hipMemcpy failed on %s size of %d bytes\n", #a, (int)(b * sizeof(c))) ; \
        fprintf (stderr, "Error: %s = %d, %s\n", #d, d, hipGetErrorString (d)) ; \
        return (E_NOMEM) ; \
    }

extern "C"
__global__ void cuMUTload_kernel (MUTparamGPUstruct, double *, double *, double *, int, double, double, int, int, int *, double *, int *, double *) ;

extern "C"
int
cuMUTload
(
GENmodel *inModel, CKTcircuit *ckt
)
{
    MUTmodel *model = (MUTmodel *)inModel ;
    int thread_x, thread_y, block_x ;

    hipError_t status ;

    /*  loop through all the mutual inductor models */
    for ( ; model != NULL ; model = model->MUTnextModel)
    {
        /* Determining how many blocks should exist in the kernel */
        thread_x = 1 ;
        thread_y = 256 ;
        if (model->n_instances % thread_y != 0)
            block_x = (int)(model->n_instances / thread_y) + 1 ;
        else
            block_x = model->n_instances / thread_y ;

        dim3 thread (thread_x, thread_y) ;

        /* Kernel launch */
        status = hipGetLastError () ; // clear error status

        cuMUTload_kernel <<< block_x, thread >>> (model->MUTparamGPU, ckt->d_CKTrhsOld, ckt->d_CKTstate0,
                                                  ckt->d_CKTstate1, ckt->CKTmode, ckt->CKTag [0], ckt->CKTag [1],
                                                  ckt->CKTorder, model->n_instances,
                                                  model->d_PositionVector, ckt->d_CKTloadOutput,
                                                  model->d_PositionVectorRHS, ckt->d_CKTloadOutputRHS) ;

        hipDeviceSynchronize () ;

        status = hipGetLastError () ; // check for launch error
        if (status != hipSuccess)
        {
            fprintf (stderr, "Kernel launch failure in the Mutual Inductor Model\n\n") ;
            return (E_NOMEM) ;
        }
    }

    return (OK) ;
}

extern "C"
__global__
void
cuMUTload_kernel
(
MUTparamGPUstruct MUTentry, double *CKTrhsOld, double *CKTstate_0,
double *CKTstate_1, int CKTmode, double CKTag_0, double CKTag_1,
int CKTorder, int mut_n_instances,
int *d_PositionVector, double *d_CKTloadOutput,
int *d_PositionVectorRHS, double *d_CKTloadOutputRHS
)
{
    int instance_ID ;
    int error ;
    double req_dummy, veq ;

    instance_ID = threadIdx.y + blockDim.y * blockIdx.x ;

    if (instance_ID < mut_n_instances)
    {
        if (threadIdx.x == 0)
        {
            if (!(CKTmode & (MODEDC | MODEINITPRED)))
            {
                CKTstate_0 [MUTentry.d_MUTflux1Array [instance_ID]] += MUTentry.d_MUTfactorArray [instance_ID] * CKTrhsOld [MUTentry.d_MUTbrEq2Array [instance_ID]] ;
                CKTstate_0 [MUTentry.d_MUTflux2Array [instance_ID]] += MUTentry.d_MUTfactorArray [instance_ID] * CKTrhsOld [MUTentry.d_MUTbrEq1Array [instance_ID]] ;
            }

            /* Inductor-related */
            if (CKTmode & MODEINITTRAN)
            {
                CKTstate_1 [MUTentry.d_MUTflux1Array [instance_ID]] = CKTstate_0 [MUTentry.d_MUTflux1Array [instance_ID]] ;
                CKTstate_1 [MUTentry.d_MUTflux2Array [instance_ID]] = CKTstate_0 [MUTentry.d_MUTflux2Array [instance_ID]] ;
            }

            if (!(CKTmode & MODEDC))
            {
                error = cuNIintegrate_device_kernel (CKTstate_0, CKTstate_1, &req_dummy, &veq,
                                                    1.0, MUTentry.d_MUTflux1Array [instance_ID],
                                                    CKTag_0, CKTag_1, CKTorder) ;
                if (error)
                    printf ("Error in the integration 1 of MUTload!\n\n") ;

                /* Output for the RHS */
                d_CKTloadOutputRHS [d_PositionVectorRHS [MUTentry.d_MUTinstanceIND1Array [instance_ID]]] = veq ;


                error = cuNIintegrate_device_kernel (CKTstate_0, CKTstate_1, &req_dummy, &veq,
                                                    1.0, MUTentry.d_MUTflux2Array [instance_ID],
                                                    CKTag_0, CKTag_1, CKTorder) ;
                if (error)
                    printf ("Error in the integration 2 of MUTload!\n\n") ;

                /* Output for the RHS */
                d_CKTloadOutputRHS [d_PositionVectorRHS [MUTentry.d_MUTinstanceIND2Array [instance_ID]]] = veq ;
            }

            if (CKTmode & MODEINITTRAN)
            {
                CKTstate_1 [MUTentry.d_MUTflux1Array [instance_ID] + 1] = CKTstate_0 [MUTentry.d_MUTflux1Array [instance_ID] + 1] ;
                CKTstate_1 [MUTentry.d_MUTflux2Array [instance_ID] + 1] = CKTstate_0 [MUTentry.d_MUTflux2Array [instance_ID] + 1] ;

            }

            d_CKTloadOutput [d_PositionVector [instance_ID]] = MUTentry.d_MUTfactorArray [instance_ID] * CKTag_0 ;
        }
    }

    return ;
}
